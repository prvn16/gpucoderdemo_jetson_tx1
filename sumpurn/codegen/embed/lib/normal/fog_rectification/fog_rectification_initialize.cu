//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: fog_rectification_initialize.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 25-Jan-2018 08:58:04
//

// Include Files
#include "rt_nonfinite.h"
#include "fog_rectification.h"
#include "fog_rectification_initialize.h"

// Function Definitions

//
// Arguments    : void
// Return Type  : void
//
void fog_rectification_initialize()
{
  rt_InitInfAndNaN(8U);
}

//
// File trailer for fog_rectification_initialize.cu
//
// [EOF]
//

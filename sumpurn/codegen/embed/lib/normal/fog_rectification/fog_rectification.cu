#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: fog_rectification.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 25-Jan-2018 08:58:04
//

// Include Files
#include "rt_nonfinite.h"
#include "fog_rectification.h"

// Variable Definitions
__constant__ real_T const_b[9];

// Function Declarations
static __global__ void fog_rectification_kernel1(const uint8_T *input, real_T
  *b_input);
static __global__ void fog_rectification_kernel10(real_T *restoreOut, uint8_T
  *b_restoreOut);
static __global__ void fog_rectification_kernel11(const real_T *b, uint8_T
  *restoreOut, uint8_T *im_gray);
static __global__ void fog_rectification_kernel12(real_T *localBins3, real_T
  *localBins2, real_T *localBins1, real_T *cdf);
static __global__ void fog_rectification_kernel13(real_T *localBins3, real_T
  *localBins2, real_T *localBins1, real_T *cdf);
static __global__ void fog_rectification_kernel14(int32_T *y_size, int32_T
  *b_y_size, int32_T *ii_size, int32_T *T_size);
static __global__ void fog_rectification_kernel15(int32_T i, real_T *T_data);
static __global__ void fog_rectification_kernel16(uint8_T *restoreOut);
static __global__ void fog_rectification_kernel17(real_T *T_data, uint8_T
  *restoreOut, uint8_T *out);
static __global__ void fog_rectification_kernel18(real_T *T_data, uint8_T
  *restoreOut, uint8_T *out);
static __global__ void fog_rectification_kernel19(real_T *T_data, uint8_T
  *restoreOut, uint8_T *out);
static __global__ void fog_rectification_kernel2(real_T *input, real_T
  *darkChannel);
static __global__ void fog_rectification_kernel3(real_T *darkChannel, real_T
  *diff_im);
static __global__ void fog_rectification_kernel4(real_T *expanded);
static __global__ void fog_rectification_kernel5(real_T *diff_im, real_T
  *expanded);
static __global__ void fog_rectification_kernel6(real_T *expanded, real_T
  *diff_im);
static __global__ void fog_rectification_kernel7(real_T *diff_im, real_T *y);
static __global__ void fog_rectification_kernel8(real_T *y, real_T *diff_im,
  real_T *darkChannel);
static __global__ void fog_rectification_kernel9(real_T *darkChannel, real_T
  *diff_im, real_T *input, real_T *restoreOut);
static __device__ real_T rt_roundd_snf(real_T u);

// Function Definitions

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                const uint8_T *input
//                real_T *b_input
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel1(const
  uint8_T *input, real_T *b_input)
{
  int32_T j;
  ;
  ;
  j = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(int32_T)(j >= 921600)) {
    //  restoreOut is used to store the output of restoration
    //  Changing the precision level of input image to double
    b_input[j] = (real_T)input[j] / 255.0;
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *restoreOut
//                uint8_T *b_restoreOut
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel10
  (real_T *restoreOut, uint8_T *b_restoreOut)
{
  real_T cv;
  int32_T j;
  uint8_T u0;
  ;
  ;
  j = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(int32_T)(j >= 921600)) {
    cv = rt_roundd_snf(255.0 * restoreOut[j]);
    if (cv < 256.0) {
      if (cv >= 0.0) {
        u0 = (uint8_T)cv;
      } else {
        u0 = 0;
      }
    } else if (cv >= 256.0) {
      u0 = MAX_uint8_T;
    } else {
      u0 = 0;
    }

    b_restoreOut[j] = u0;
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                const real_T *b
//                uint8_T *restoreOut
//                uint8_T *im_gray
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel11(
  const real_T *b, uint8_T *restoreOut, uint8_T *im_gray)
{
  uint8_T a[3];
  int32_T j;
  real_T cv;
  int32_T n;
  uint8_T u0;
  ;
  ;
  j = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(int32_T)(j >= 307200)) {
    // %%%%%% streching performs the histogram streching of the image %%%%%%%
    // %%%%%%%% im is the input color image and p is cdf limit
    // %%%%% out is the contrast streched image and cdf is the cumulative prob
    // %%%%% density function and T is the streching function
    //  rgbtograyconversion
    a[0] = restoreOut[j];
    a[1] = restoreOut[j + 307200];
    a[2] = restoreOut[j + 614400];
    cv = 0.0;
    for (n = 0; n < 3; n++) {
      cv += (real_T)a[n] * b[n];
    }

    cv = rt_roundd_snf(cv);
    if (cv < 256.0) {
      u0 = (uint8_T)cv;
    } else {
      u0 = MAX_uint8_T;
    }

    im_gray[j] = u0;
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *localBins3
//                real_T *localBins2
//                real_T *localBins1
//                real_T *cdf
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void fog_rectification_kernel12
  (real_T *localBins3, real_T *localBins2, real_T *localBins1, real_T *cdf)
{
  int32_T j;
  ;
  ;
  j = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(int32_T)(j >= 256)) {
    //  histogram calculation
    cdf[j] = 0.0;
    localBins1[j] = 0.0;
    localBins2[j] = 0.0;
    localBins3[j] = 0.0;
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *localBins3
//                real_T *localBins2
//                real_T *localBins1
//                real_T *cdf
// Return Type  : void
//
static __global__ __launch_bounds__(256, 1) void fog_rectification_kernel13
  (real_T *localBins3, real_T *localBins2, real_T *localBins1, real_T *cdf)
{
  int32_T j;
  ;
  ;
  j = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(int32_T)(j >= 256)) {
    //  cumulative Sum calculation
    cdf[j] = ((cdf[j] + localBins1[j]) + localBins2[j]) + localBins3[j];
    cdf[j] /= 307200.0;
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                int32_T *y_size
//                int32_T *b_y_size
//                int32_T *ii_size
//                int32_T *T_size
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void fog_rectification_kernel14
  (int32_T *y_size, int32_T *b_y_size, int32_T *ii_size, int32_T *T_size)
{
  ;
  ;
  if (!(int32_T)((int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x *
            blockIdx.y) + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
          threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x) +
        threadIdx.x) >= 1)) {
    T_size[0] = 1;
    T_size[1] = ((ii_size[0] + b_y_size[1]) + y_size[1]) + 1;
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                int32_T i
//                real_T *T_data
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel15
  (int32_T i, real_T *T_data)
{
  int32_T n;
  ;
  ;
  n = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if ((!(int32_T)(n >= 768)) && ((int32_T)(1 + n <= i))) {
    T_data[n] = floor(T_data[n]);
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                uint8_T *restoreOut
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel16
  (uint8_T *restoreOut)
{
  int32_T j;
  ;
  ;
  j = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if ((!(int32_T)(j >= 921600)) && ((int32_T)((int32_T)restoreOut[j] == 0))) {
    //  Replacing the value from look up table
    restoreOut[j] = 1;
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *T_data
//                uint8_T *restoreOut
//                uint8_T *out
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel17
  (real_T *T_data, uint8_T *restoreOut, uint8_T *out)
{
  uint32_T threadId;
  real_T cv;
  int32_T j;
  int32_T i0;
  uint8_T u0;
  ;
  ;
  threadId = ((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y) +
                blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
               threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x)
    + threadIdx.x;
  i0 = (int32_T)(threadId / 480U);
  j = (int32_T)(threadId - (uint32_T)i0 * 480U);
  if ((!(int32_T)(j >= 480)) && (!(int32_T)(i0 >= 640))) {
    cv = rt_roundd_snf(T_data[(int32_T)restoreOut[j + 480 * i0] - 1]);
    if (cv < 256.0) {
      if (cv >= 0.0) {
        u0 = (uint8_T)cv;
      } else {
        u0 = 0;
      }
    } else if (cv >= 256.0) {
      u0 = MAX_uint8_T;
    } else {
      u0 = 0;
    }

    out[j + 480 * i0] = u0;
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *T_data
//                uint8_T *restoreOut
//                uint8_T *out
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel18
  (real_T *T_data, uint8_T *restoreOut, uint8_T *out)
{
  uint32_T threadId;
  real_T cv;
  int32_T j;
  int32_T i0;
  uint8_T u0;
  ;
  ;
  threadId = ((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y) +
                blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
               threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x)
    + threadIdx.x;
  i0 = (int32_T)(threadId / 480U);
  j = (int32_T)(threadId - (uint32_T)i0 * 480U);
  if ((!(int32_T)(j >= 480)) && (!(int32_T)(i0 >= 640))) {
    cv = rt_roundd_snf(T_data[(int32_T)restoreOut[307200 + (j + 480 * i0)] - 1]);
    if (cv < 256.0) {
      if (cv >= 0.0) {
        u0 = (uint8_T)cv;
      } else {
        u0 = 0;
      }
    } else if (cv >= 256.0) {
      u0 = MAX_uint8_T;
    } else {
      u0 = 0;
    }

    out[307200 + (j + 480 * i0)] = u0;
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *T_data
//                uint8_T *restoreOut
//                uint8_T *out
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel19
  (real_T *T_data, uint8_T *restoreOut, uint8_T *out)
{
  uint32_T threadId;
  real_T cv;
  int32_T j;
  int32_T i0;
  uint8_T u0;
  ;
  ;
  threadId = ((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y) +
                blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
               threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x)
    + threadIdx.x;
  i0 = (int32_T)(threadId / 480U);
  j = (int32_T)(threadId - (uint32_T)i0 * 480U);
  if ((!(int32_T)(j >= 480)) && (!(int32_T)(i0 >= 640))) {
    cv = rt_roundd_snf(T_data[(int32_T)restoreOut[614400 + (j + 480 * i0)] - 1]);
    if (cv < 256.0) {
      if (cv >= 0.0) {
        u0 = (uint8_T)cv;
      } else {
        u0 = 0;
      }
    } else if (cv >= 256.0) {
      u0 = MAX_uint8_T;
    } else {
      u0 = 0;
    }

    out[614400 + (j + 480 * i0)] = u0;
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *input
//                real_T *darkChannel
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel2
  (real_T *input, real_T *darkChannel)
{
  real_T cv;
  int32_T j;
  int32_T n;
  ;
  ;
  j = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(int32_T)(j >= 307200)) {
    //  Dark channel Estimation from input
    cv = input[j];
    for (n = j + 307201; n <= j + 614401; n += 307200) {
      if (input[n - 1] < cv) {
        cv = input[n - 1];
      }
    }

    darkChannel[j] = cv;
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *darkChannel
//                real_T *diff_im
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel3
  (real_T *darkChannel, real_T *diff_im)
{
  int32_T j;
  ;
  ;
  j = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(int32_T)(j >= 307200)) {
    //  diff_im is used as input and output variable for anisotropic diffusion
    diff_im[j] = 0.9 * darkChannel[j];
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *expanded
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel4
  (real_T *expanded)
{
  int32_T j;
  ;
  ;
  j = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(int32_T)(j >= 309444)) {
    expanded[j] = 0.0;
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *diff_im
//                real_T *expanded
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel5
  (real_T *diff_im, real_T *expanded)
{
  uint32_T threadId;
  int32_T j;
  int32_T i0;
  ;
  ;
  threadId = ((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y) +
                blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
               threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x)
    + threadIdx.x;
  i0 = (int32_T)(threadId / 480U);
  j = (int32_T)(threadId - (uint32_T)i0 * 480U);
  if ((!(int32_T)(j >= 480)) && (!(int32_T)(i0 >= 640))) {
    expanded[(j + 482 * (1 + i0)) + 1] = diff_im[j + 480 * i0];
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *expanded
//                real_T *diff_im
// Return Type  : void
//
static __global__ __launch_bounds__(1024, 1) void fog_rectification_kernel6
  (real_T *expanded, real_T *diff_im)
{
  real_T cv;
  int32_T n;
  int32_T j;
  int32_T threadIdY;
  int32_T threadIdX;
  __shared__ real_T expanded_shared[1156];
  int32_T baseR;
  int32_T srow;
  int32_T strideRow;
  int32_T scol;
  int32_T strideCol;
  int32_T y_idx;
  int32_T baseC;
  int32_T x_idx;
  ;
  ;
  threadIdY = (int32_T)(blockDim.y * blockIdx.y + threadIdx.y);
  threadIdX = (int32_T)(blockDim.x * blockIdx.x + threadIdx.x);
  baseR = threadIdX;
  srow = (int32_T)threadIdx.x;
  strideRow = (int32_T)blockDim.x;
  scol = (int32_T)threadIdx.y;
  strideCol = (int32_T)blockDim.y;
  for (y_idx = srow; y_idx <= 33; y_idx += strideRow) {
    baseC = threadIdY;
    for (x_idx = scol; x_idx <= 33; x_idx += strideCol) {
      if (((int32_T)(((int32_T)(baseR >= 0)) && ((int32_T)(baseR < 482)))) &&
          ((int32_T)(((int32_T)(baseC >= 0)) && ((int32_T)(baseC < 642))))) {
        expanded_shared[y_idx + 34 * x_idx] = (real_T)expanded[482 * baseC +
          baseR];
      } else {
        expanded_shared[y_idx + 34 * x_idx] = 0.0;
      }

      baseC += strideCol;
    }

    baseR += strideRow;
  }

  __syncthreads();
  if ((!(int32_T)(threadIdX >= 480)) && (!(int32_T)(threadIdY >= 640))) {
    cv = 0.0;
    for (n = 0; n < 3; n++) {
      for (j = 0; j < 3; j++) {
        cv += expanded_shared[((int32_T)threadIdx.x + ((j + threadIdX) -
          threadIdX)) + 34 * ((int32_T)threadIdx.y + ((n + threadIdY) -
          threadIdY))] * const_b[(3 * (2 - n) - j) + 2];
      }
    }

    diff_im[threadIdX + 480 * threadIdY] = cv;
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *diff_im
//                real_T *y
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel7
  (real_T *diff_im, real_T *y)
{
  int32_T j;
  ;
  ;
  j = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(int32_T)(j >= 307200)) {
    //  Reduction with min
    y[j] = diff_im[j];
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *y
//                real_T *diff_im
//                real_T *darkChannel
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel8
  (real_T *y, real_T *diff_im, real_T *darkChannel)
{
  real_T u1;
  int32_T n;
  ;
  ;
  n = (int32_T)(((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y)
                   + blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
                  threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y *
                 blockDim.x) + threadIdx.x);
  if (!(int32_T)(n >= 307200)) {
    //  Parallel element-wise math to compute
    //   Restoration with inverse Koschmieder's law
    u1 = y[n];
    if (darkChannel[n] < y[n]) {
      u1 = darkChannel[n];
    }

    diff_im[n] = u1;
    diff_im[n] *= 0.6;
    darkChannel[n] = 1.0 / (1.0 - diff_im[n]);
  }
}

//
// Arguments    : uint3 blockArg
//                uint3 gridArg
//                real_T *darkChannel
//                real_T *diff_im
//                real_T *input
//                real_T *restoreOut
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void fog_rectification_kernel9
  (real_T *darkChannel, real_T *diff_im, real_T *input, real_T *restoreOut)
{
  uint32_T threadId;
  int32_T j;
  int32_T i0;
  ;
  ;
  threadId = ((((gridDim.x * gridDim.y * blockIdx.z + gridDim.x * blockIdx.y) +
                blockIdx.x) * (blockDim.x * blockDim.y * blockDim.z) +
               threadIdx.z * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x)
    + threadIdx.x;
  i0 = (int32_T)(threadId / 480U);
  j = (int32_T)(threadId - (uint32_T)i0 * 480U);
  if ((!(int32_T)(j >= 480)) && (!(int32_T)(i0 >= 640))) {
    restoreOut[j + 480 * i0] = (input[j + 480 * i0] - diff_im[j + 480 * i0]) *
      darkChannel[j + 480 * i0];
    restoreOut[307200 + (j + 480 * i0)] = (input[307200 + (j + 480 * i0)] -
      diff_im[j + 480 * i0]) * darkChannel[j + 480 * i0];
    restoreOut[614400 + (j + 480 * i0)] = (input[614400 + (j + 480 * i0)] -
      diff_im[j + 480 * i0]) * darkChannel[j + 480 * i0];
  }
}

//
// Arguments    : real_T u
// Return Type  : real_T
//
static __device__ real_T rt_roundd_snf(real_T u)
{
  real_T y;
  if (fabs(u) < 4.503599627370496E+15) {
    if (u >= 0.5) {
      y = floor(u + 0.5);
    } else if (u > -0.5) {
      y = u * 0.0;
    } else {
      y = ceil(u - 0.5);
    }
  } else {
    y = u;
  }

  return y;
}

//
// Copyright 2017 The MathWorks, Inc.
// Arguments    : const uint8_T input[921600]
//                uint8_T out[921600]
// Return Type  : void
//
void fog_rectification(const uint8_T input[921600], uint8_T out[921600])
{
  int32_T idx;
  int32_T i0;
  int32_T i;
  static const real_T b[9] = { 0.0625, 0.125, 0.0625, 0.125, 0.25, 0.125, 0.0625,
    0.125, 0.0625 };

  static uint8_T im_gray[307200];
  static const real_T b_b[3] = { 0.29893602129377539, 0.58704307445112125,
    0.11402090425510336 };

  real_T cdf[256];
  real_T localBins1[256];
  real_T localBins2[256];
  real_T localBins3[256];
  int32_T ii_size[1];
  int32_T varargin_1;
  int32_T b_ii_size[1];
  real_T y;
  real_T b_y;
  real_T y_data[255];
  int32_T y_size[2];
  int32_T ndbl;
  int16_T i1;
  int16_T i2;
  real_T c_y;
  int32_T absb;
  int32_T u0;
  uint32_T u1;
  int32_T b_y_size[2];
  real_T d_y;
  real_T e_y;
  int32_T T_size[2];
  real_T T_data[771];
  uint8_T *gpu_input;
  real_T *b_gpu_input;
  real_T *gpu_darkChannel;
  real_T *gpu_diff_im;
  real_T *gpu_expanded;
  real_T *gpu_y;
  real_T *gpu_restoreOut;
  uint8_T *b_gpu_restoreOut;
  real_T *gpu_b;
  uint8_T *gpu_im_gray;
  real_T *gpu_localBins3;
  real_T *gpu_localBins2;
  real_T *gpu_localBins1;
  real_T *gpu_cdf;
  int32_T *gpu_y_size;
  int32_T *b_gpu_y_size;
  int32_T *gpu_ii_size;
  int32_T *gpu_T_size;
  real_T *gpu_T_data;
  uint8_T *gpu_out;
  boolean_T im_gray_dirtyOnGpu;
  boolean_T localBins3_dirtyOnGpu;
  boolean_T localBins2_dirtyOnGpu;
  boolean_T localBins1_dirtyOnGpu;
  boolean_T cdf_dirtyOnGpu;
  boolean_T T_size_dirtyOnGpu;
  boolean_T localBins3_dirtyOnCpu;
  boolean_T localBins2_dirtyOnCpu;
  boolean_T localBins1_dirtyOnCpu;
  boolean_T cdf_dirtyOnCpu;
  boolean_T T_data_dirtyOnCpu;
  boolean_T exitg1;
  hipMalloc(&gpu_out, 921600ULL);
  hipMalloc(&gpu_T_data, 771U * sizeof(real_T));
  hipMalloc(&gpu_T_size, 8ULL);
  hipMalloc(&gpu_y_size, 8ULL);
  hipMalloc(&b_gpu_y_size, 8ULL);
  hipMalloc(&gpu_ii_size, 4ULL);
  hipMalloc(&gpu_localBins1, 2048ULL);
  hipMalloc(&gpu_localBins2, 2048ULL);
  hipMalloc(&gpu_localBins3, 2048ULL);
  hipMalloc(&gpu_cdf, 2048ULL);
  hipMalloc(&gpu_im_gray, 307200ULL);
  hipMalloc(&gpu_b, 24ULL);
  hipMalloc(&b_gpu_restoreOut, 921600ULL);
  hipMalloc(&gpu_restoreOut, 7372800ULL);
  hipMalloc(&gpu_y, 2457600ULL);
  hipMalloc(&gpu_diff_im, 2457600ULL);
  hipMalloc(&gpu_expanded, 2475552ULL);
  hipMalloc(&gpu_darkChannel, 2457600ULL);
  hipMalloc(&b_gpu_input, 7372800ULL);
  hipMalloc(&gpu_input, 921600ULL);
  T_data_dirtyOnCpu = false;
  cdf_dirtyOnCpu = false;
  localBins1_dirtyOnCpu = false;
  localBins2_dirtyOnCpu = false;
  localBins3_dirtyOnCpu = false;

  //  restoreOut is used to store the output of restoration
  //  Changing the precision level of input image to double
  hipMemcpy((void *)gpu_input, (void *)&input[0], 921600ULL,
             hipMemcpyHostToDevice);
  fog_rectification_kernel1<<<dim3(1800U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_input, b_gpu_input);

  //  Dark channel Estimation from input
  fog_rectification_kernel2<<<dim3(600U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (b_gpu_input, gpu_darkChannel);

  //  diff_im is used as input and output variable for anisotropic diffusion
  fog_rectification_kernel3<<<dim3(600U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_darkChannel, gpu_diff_im);

  //  2D convolution mask for Anisotropic diffusion
  //  Refine dark channel using Anisotropic diffusion.
  for (idx = 0; idx < 3; idx++) {
    fog_rectification_kernel4<<<dim3(605U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (gpu_expanded);
    fog_rectification_kernel5<<<dim3(600U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (gpu_diff_im, gpu_expanded);
    hipMemcpyToSymbol(HIP_SYMBOL(const_b), b, 72ULL, 0ULL, hipMemcpyHostToDevice);
    fog_rectification_kernel6<<<dim3(15U, 20U, 1U), dim3(32U, 32U, 1U)>>>
      (gpu_expanded, gpu_diff_im);
  }

  //  Reduction with min
  fog_rectification_kernel7<<<dim3(600U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_diff_im, gpu_y);

  //  Parallel element-wise math to compute
  //   Restoration with inverse Koschmieder's law
  fog_rectification_kernel8<<<dim3(600U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_y,
    gpu_diff_im, gpu_darkChannel);
  fog_rectification_kernel9<<<dim3(600U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_darkChannel, gpu_diff_im, b_gpu_input, gpu_restoreOut);
  fog_rectification_kernel10<<<dim3(1800U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_restoreOut, b_gpu_restoreOut);

  // %%%%%% streching performs the histogram streching of the image %%%%%%%
  // %%%%%%%% im is the input color image and p is cdf limit
  // %%%%% out is the contrast streched image and cdf is the cumulative prob
  // %%%%% density function and T is the streching function
  //  rgbtograyconversion
  hipMemcpy((void *)gpu_b, (void *)&b_b[0], 24ULL, hipMemcpyHostToDevice);
  fog_rectification_kernel11<<<dim3(600U, 1U, 1U), dim3(512U, 1U, 1U)>>>(gpu_b,
    b_gpu_restoreOut, gpu_im_gray);
  im_gray_dirtyOnGpu = true;

  //  histogram calculation
  fog_rectification_kernel12<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>
    (gpu_localBins3, gpu_localBins2, gpu_localBins1, gpu_cdf);
  cdf_dirtyOnGpu = true;
  localBins1_dirtyOnGpu = true;
  localBins2_dirtyOnGpu = true;
  localBins3_dirtyOnGpu = true;
  for (i = 1; i + 3 <= 307200; i += 4) {
    if (im_gray_dirtyOnGpu) {
      hipMemcpy((void *)&im_gray[0], (void *)gpu_im_gray, 307200ULL,
                 hipMemcpyDeviceToHost);
      im_gray_dirtyOnGpu = false;
    }

    if (localBins1_dirtyOnGpu) {
      hipMemcpy((void *)&localBins1[0], (void *)gpu_localBins1, 2048ULL,
                 hipMemcpyDeviceToHost);
      localBins1_dirtyOnGpu = false;
    }

    localBins1[im_gray[i - 1]]++;
    localBins1_dirtyOnCpu = true;
    if (localBins2_dirtyOnGpu) {
      hipMemcpy((void *)&localBins2[0], (void *)gpu_localBins2, 2048ULL,
                 hipMemcpyDeviceToHost);
      localBins2_dirtyOnGpu = false;
    }

    localBins2[im_gray[i]]++;
    localBins2_dirtyOnCpu = true;
    if (localBins3_dirtyOnGpu) {
      hipMemcpy((void *)&localBins3[0], (void *)gpu_localBins3, 2048ULL,
                 hipMemcpyDeviceToHost);
      localBins3_dirtyOnGpu = false;
    }

    localBins3[im_gray[i + 1]]++;
    localBins3_dirtyOnCpu = true;
    if (cdf_dirtyOnGpu) {
      hipMemcpy((void *)&cdf[0], (void *)gpu_cdf, 2048ULL,
                 hipMemcpyDeviceToHost);
      cdf_dirtyOnGpu = false;
    }

    cdf[im_gray[i + 2]]++;
    cdf_dirtyOnCpu = true;
  }

  for (idx = 0; idx < 307200; idx++) {
    if (1 + idx >= i) {
      if (im_gray_dirtyOnGpu) {
        hipMemcpy((void *)&im_gray[0], (void *)gpu_im_gray, 307200ULL,
                   hipMemcpyDeviceToHost);
        im_gray_dirtyOnGpu = false;
      }

      if (cdf_dirtyOnGpu) {
        hipMemcpy((void *)&cdf[0], (void *)gpu_cdf, 2048ULL,
                   hipMemcpyDeviceToHost);
        cdf_dirtyOnGpu = false;
      }

      cdf[im_gray[idx]]++;
      cdf_dirtyOnCpu = true;
    }
  }

  //  cumulative Sum calculation
  if (localBins3_dirtyOnCpu) {
    hipMemcpy((void *)gpu_localBins3, (void *)&localBins3[0], 2048ULL,
               hipMemcpyHostToDevice);
  }

  if (localBins2_dirtyOnCpu) {
    hipMemcpy((void *)gpu_localBins2, (void *)&localBins2[0], 2048ULL,
               hipMemcpyHostToDevice);
  }

  if (localBins1_dirtyOnCpu) {
    hipMemcpy((void *)gpu_localBins1, (void *)&localBins1[0], 2048ULL,
               hipMemcpyHostToDevice);
  }

  if (cdf_dirtyOnCpu) {
    hipMemcpy((void *)gpu_cdf, (void *)&cdf[0], 2048ULL, hipMemcpyHostToDevice);
  }

  fog_rectification_kernel13<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>
    (gpu_localBins3, gpu_localBins2, gpu_localBins1, gpu_cdf);
  cdf_dirtyOnGpu = true;
  for (i = 0; i < 255; i++) {
    if (cdf_dirtyOnGpu) {
      hipMemcpy((void *)&cdf[0], (void *)gpu_cdf, 2048ULL,
                 hipMemcpyDeviceToHost);
      cdf_dirtyOnGpu = false;
    }

    cdf[1 + i] += cdf[i];
  }

  //  finding less than particular probability
  idx = 0;
  ii_size[0] = 256;
  i = 1;
  exitg1 = false;
  while ((!exitg1) && (i < 257)) {
    if (cdf_dirtyOnGpu) {
      hipMemcpy((void *)&cdf[0], (void *)gpu_cdf, 2048ULL,
                 hipMemcpyDeviceToHost);
      cdf_dirtyOnGpu = false;
    }

    if (cdf[i - 1] <= 0.05) {
      idx++;
      if (idx >= 256) {
        exitg1 = true;
      } else {
        i++;
      }
    } else {
      i++;
    }
  }

  if (1 > idx) {
    varargin_1 = 0;
    ii_size[0] = 0;
  } else {
    varargin_1 = idx;
    ii_size[0] = idx;
  }

  idx = 0;
  i = 1;
  exitg1 = false;
  while ((!exitg1) && (i < 257)) {
    if (cdf_dirtyOnGpu) {
      hipMemcpy((void *)&cdf[0], (void *)gpu_cdf, 2048ULL,
                 hipMemcpyDeviceToHost);
      cdf_dirtyOnGpu = false;
    }

    if (cdf[i - 1] >= 0.95) {
      idx++;
      if (idx >= 256) {
        exitg1 = true;
      } else {
        i++;
      }
    } else {
      i++;
    }
  }

  if (1 > idx) {
    i0 = 0;
    b_ii_size[0] = 0;
  } else {
    i0 = idx;
    b_ii_size[0] = idx;
  }

  y = 25.0 / (real_T)ii_size[0];
  b_y = 204.0 / (255.0 - (real_T)(b_ii_size[0] + ii_size[0]));
  if (255 - i0 < varargin_1 + 1) {
    y_size[0] = 1;
    y_size[1] = 0;
  } else if (ii_size[0] + 1 == varargin_1 + 1) {
    i1 = (int16_T)(varargin_1 + 1);
    i2 = (int16_T)(255 - i0);
    y_size[0] = 1;
    y_size[1] = (int16_T)((int16_T)(255 - b_ii_size[0]) - (int16_T)(ii_size[0] +
      1)) + 1;
    for (i = 0; i <= (int32_T)(int16_T)(i2 - i1); i++) {
      y_data[i] = (int16_T)((int16_T)(varargin_1 + 1) + i);
    }
  } else {
    ndbl = (int32_T)std::floor((254.0 - (real_T)(b_ii_size[0] + ii_size[0])) +
      0.5);
    i = varargin_1 + ndbl;
    idx = (i + i0) - 254;
    absb = (int32_T)std::abs(255.0 - (real_T)i0);
    u0 = varargin_1 + 1;
    if (u0 > absb) {
      absb = u0;
    }

    if (std::abs((real_T)idx) < 4.4408920985006262E-16 * (real_T)absb) {
      ndbl++;
      u0 = 255 - i0;
    } else if (idx > 0) {
      u0 = varargin_1 + ndbl;
    } else {
      ndbl++;
      u0 = i + 1;
    }

    if (ndbl >= 0) {
      idx = ndbl;
    } else {
      idx = 0;
    }

    y_size[0] = 1;
    y_size[1] = idx;
    if (idx > 0) {
      y_data[0] = (real_T)varargin_1 + 1.0;
      if (idx > 1) {
        y_data[idx - 1] = u0;
        absb = (idx - 1) / 2;
        for (i = 0; i < 126; i++) {
          if (1 + i <= absb - 1) {
            y_data[1 + i] = ((real_T)(varargin_1 + i) + 1.0) + 1.0;
            y_data[(idx - i) - 2] = (u0 - i) - 1;
          }
        }

        if (absb << 1 == idx - 1) {
          y_data[absb] = ((real_T)(varargin_1 + u0) + 1.0) / 2.0;
        } else {
          y_data[absb] = (real_T)(varargin_1 + absb) + 1.0;
          y_data[absb + 1] = u0 - absb;
        }
      }
    }
  }

  c_y = 26.0 / (255.0 - (255.0 - (real_T)b_ii_size[0]));
  if (255 < 256 - b_ii_size[0]) {
    b_y_size[0] = 1;
    b_y_size[1] = 0;
  } else {
    u1 = (uint32_T)((255.0 - (real_T)b_ii_size[0]) + 1.0);
    b_y_size[0] = 1;
    b_y_size[1] = (int32_T)(255.0 - ((255.0 - (real_T)i0) + 1.0)) + 1;
    for (i = 0; i <= (int32_T)(255.0 - (real_T)u1); i++) {
      if (cdf_dirtyOnGpu) {
        hipMemcpy((void *)&cdf[0], (void *)gpu_cdf, 2048ULL,
                   hipMemcpyDeviceToHost);
        cdf_dirtyOnGpu = false;
      }

      cdf[i] = ((255.0 - (real_T)i0) + 1.0) + (real_T)i;
    }
  }

  d_y = 204.0 / (255.0 - (real_T)(i0 + varargin_1)) * (real_T)varargin_1;
  e_y = 26.0 / (255.0 - (255.0 - (real_T)i0)) * (255.0 - (real_T)b_ii_size[0]);
  hipMemcpy((void *)gpu_y_size, (void *)&b_y_size[0], 8ULL,
             hipMemcpyHostToDevice);
  hipMemcpy((void *)b_gpu_y_size, (void *)&y_size[0], 8ULL,
             hipMemcpyHostToDevice);
  hipMemcpy((void *)gpu_ii_size, (void *)&ii_size[0], 4ULL,
             hipMemcpyHostToDevice);
  fog_rectification_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_y_size,
    b_gpu_y_size, gpu_ii_size, gpu_T_size);
  T_size_dirtyOnGpu = true;
  for (i0 = 0; i0 <= varargin_1; i0++) {
    if (T_size_dirtyOnGpu) {
      hipMemcpy((void *)&T_size[0], (void *)gpu_T_size, 8ULL,
                 hipMemcpyDeviceToHost);
      T_size_dirtyOnGpu = false;
    }

    T_data[i0] = y * (real_T)i0;
    T_data_dirtyOnCpu = true;
  }

  i = y_size[1];
  for (i0 = 0; i0 < i; i0++) {
    if (T_size_dirtyOnGpu) {
      hipMemcpy((void *)&T_size[0], (void *)gpu_T_size, 8ULL,
                 hipMemcpyDeviceToHost);
      T_size_dirtyOnGpu = false;
    }

    T_data[(i0 + varargin_1) + 1] = (b_y * y_data[i0] - d_y) + 25.0;
    T_data_dirtyOnCpu = true;
  }

  i = b_y_size[1];
  for (i0 = 0; i0 < i; i0++) {
    if (T_size_dirtyOnGpu) {
      hipMemcpy((void *)&T_size[0], (void *)gpu_T_size, 8ULL,
                 hipMemcpyDeviceToHost);
      T_size_dirtyOnGpu = false;
    }

    if (cdf_dirtyOnGpu) {
      hipMemcpy((void *)&cdf[0], (void *)gpu_cdf, 2048ULL,
                 hipMemcpyDeviceToHost);
      cdf_dirtyOnGpu = false;
    }

    T_data[((i0 + varargin_1) + y_size[1]) + 1] = (c_y * cdf[i0] - e_y) + 229.0;
    T_data_dirtyOnCpu = true;
  }

  if (T_size_dirtyOnGpu) {
    hipMemcpy((void *)&T_size[0], (void *)gpu_T_size, 8ULL,
               hipMemcpyDeviceToHost);
  }

  i = T_size[1];
  if (T_data_dirtyOnCpu) {
    hipMemcpy((void *)gpu_T_data, (void *)&T_data[0], T_size[0] * T_size[1] *
               sizeof(real_T), hipMemcpyHostToDevice);
  }

  fog_rectification_kernel15<<<dim3(2U, 1U, 1U), dim3(512U, 1U, 1U)>>>(i,
    gpu_T_data);

  //  Replacing the value from look up table
  fog_rectification_kernel16<<<dim3(1800U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (b_gpu_restoreOut);
  fog_rectification_kernel17<<<dim3(600U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_T_data, b_gpu_restoreOut, gpu_out);
  fog_rectification_kernel18<<<dim3(600U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_T_data, b_gpu_restoreOut, gpu_out);
  fog_rectification_kernel19<<<dim3(600U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (gpu_T_data, b_gpu_restoreOut, gpu_out);
  hipMemcpy((void *)&out[0], (void *)gpu_out, 921600ULL, hipMemcpyDeviceToHost);
  hipFree(gpu_input);
  hipFree(b_gpu_input);
  hipFree(gpu_darkChannel);
  hipFree(gpu_expanded);
  hipFree(gpu_diff_im);
  hipFree(gpu_y);
  hipFree(gpu_restoreOut);
  hipFree(b_gpu_restoreOut);
  hipFree(gpu_b);
  hipFree(gpu_im_gray);
  hipFree(gpu_cdf);
  hipFree(gpu_localBins3);
  hipFree(gpu_localBins2);
  hipFree(gpu_localBins1);
  hipFree(gpu_ii_size);
  hipFree(b_gpu_y_size);
  hipFree(gpu_y_size);
  hipFree(gpu_T_size);
  hipFree(gpu_T_data);
  hipFree(gpu_out);
}

//
// File trailer for fog_rectification.cu
//
// [EOF]
//

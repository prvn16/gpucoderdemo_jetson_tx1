//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: main.cu
//
// GPU Coder version                    : 1.0
// CUDA/C/C++ source code generated on  : 25-Jan-2018 08:58:04
//

//***********************************************************************
// This automatically generated example C main file shows how to call
// entry-point functions that MATLAB Coder generated. You must customize
// this file for your application. Do not modify this file directly.
// Instead, make a copy of this file, modify it, and integrate it into
// your development environment.
//
// This file initializes entry-point function arguments to a default
// size and value before calling the entry-point functions. It does
// not store or use any values returned from the entry-point functions.
// If necessary, it does pre-allocate memory for returned values.
// You can use this file as a starting point for a main function that
// you can deploy in your application.
//
// After you copy the file, and before you deploy it, you must make the
// following changes:
// * For variable-size function arguments, change the example sizes to
// the sizes that your application requires.
// * Change the example values of function arguments to the values that
// your application requires.
// * If the entry-point functions return values, store these values or
// otherwise use them as required by your application.
//
//***********************************************************************
// Include Files
#include "rt_nonfinite.h"
#include "fog_rectification.h"
#include "main.h"
#include "fog_rectification_terminate.h"
#include "fog_rectification_initialize.h"

// Function Declarations
static void argInit_480x640x3_uint8_T(uint8_T result[921600]);
static uint8_T argInit_uint8_T();
static void main_fog_rectification();

// Function Definitions

//
// Arguments    : uint8_T result[921600]
// Return Type  : void
//
static void argInit_480x640x3_uint8_T(uint8_T result[921600])
{
  int32_T idx0;
  int32_T idx1;
  int32_T idx2;

  // Loop over the array to initialize each element.
  for (idx0 = 0; idx0 < 480; idx0++) {
    for (idx1 = 0; idx1 < 640; idx1++) {
      for (idx2 = 0; idx2 < 3; idx2++) {
        // Set the value of the array element.
        // Change this value to the value that the application requires.
        result[(idx0 + 480 * idx1) + 307200 * idx2] = argInit_uint8_T();
      }
    }
  }
}

//
// Arguments    : void
// Return Type  : uint8_T
//
static uint8_T argInit_uint8_T()
{
  return 0;
}

//
// Arguments    : void
// Return Type  : void
//
static void main_fog_rectification()
{
  static uint8_T out[921600];
  static uint8_T b[921600];

  // Initialize function 'fog_rectification' input arguments.
  // Initialize function input argument 'input'.
  // Call the entry-point 'fog_rectification'.
  argInit_480x640x3_uint8_T(b);
  fog_rectification(b, out);
}

//
// Arguments    : int32_T argc
//                const char * const argv[]
// Return Type  : int32_T
//
int32_T main(int32_T, const char * const [])
{
  // Initialize the application.
  // You do not need to do this more than one time.
  fog_rectification_initialize();

  // Invoke the entry-point functions.
  // You can call entry-point functions multiple times.
  main_fog_rectification();

  // Terminate the application.
  // You do not need to do this more than one time.
  fog_rectification_terminate();
  return 0;
}

//
// File trailer for main.cu
//
// [EOF]
//
